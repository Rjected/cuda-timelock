#include "hip/hip_runtime.h"
/***
Copyright (c) 2018-2019, NVIDIA CORPORATION.  All rights reserved.
Permission is hereby granted, free of charge, to any person obtaining a
copy of this software and associated documentation files (the "Software"),
to deal in the Software without restriction, including without limitation
the rights to use, copy, modify, merge, publish, distribute, sublicense,
and/or sell copies of the Software, and to permit persons to whom the
Software is furnished to do so, subject to the following conditions:
The above copyright notice and this permission notice shall be included in
all copies or substantial portions of the Software.
THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
IN THE SOFTWARE.
***/

#include <time.h>
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <gmp.h>
#include <inttypes.h>
#include "../include/cgbn/cgbn.h"
#include "../include/insecure_rsa/rsa.h"
#include "support.h"

// For this example, there are quite a few template parameters that are used to generate the actual code.
// In order to simplify passing many parameters, we use the same approach as the CGBN library, which is to
// create a container class with static constants and then pass the class.

// The CGBN context uses the following three parameters:
//   TBP             - threads per block (zero means to use the blockDim.x)
//   MAX_ROTATION    - must be small power of 2, imperically, 4 works well
//   SHM_LIMIT       - number of bytes of dynamic shared memory available to the kernel
//   CONSTANT_TIME   - require constant time algorithms (currently, constant time algorithms are not available)

// Locally it will also be helpful to have several parameters:
//   TPI             - threads per instance
//   BITS            - number of bits per instance
//   WINDOW_BITS     - number of bits to use for the windowed exponentiation

template<uint32_t tpi, uint32_t bits, uint32_t window_bits>
class powm_params_t {
  public:
  // parameters used by the CGBN context
  static const uint32_t TPB=0;                     // get TPB from blockDim.x
  static const uint32_t MAX_ROTATION=4;            // good default value
  static const uint32_t SHM_LIMIT=0;               // no shared mem available
  static const bool     CONSTANT_TIME=false;       // constant time implementations aren't available yet

  // parameters used locally in the application
  static const uint32_t TPI=tpi;                   // threads per instance
  static const uint32_t BITS=bits;                 // instance size
  static const uint32_t WINDOW_BITS=window_bits;   // window size
};

template<class params>
class powm_odd_t {
  public:
  static const uint32_t window_bits=params::WINDOW_BITS;  // used a lot, give it an instance variable

  // define the instance structure
  typedef struct {
    cgbn_mem_t<params::BITS> x;
    cgbn_mem_t<params::BITS> power;
    cgbn_mem_t<params::BITS> modulus;
    cgbn_mem_t<params::BITS> result;
  } instance_t;

  typedef cgbn_context_t<params::TPI, params>   context_t;
  typedef cgbn_env_t<context_t, params::BITS>   env_t;
  typedef typename env_t::cgbn_t                bn_t;
  typedef typename env_t::cgbn_local_t          bn_local_t;

  context_t _context;
  env_t     _env;
  int32_t   _instance;

  __device__ __forceinline__ powm_odd_t(cgbn_monitor_t monitor, cgbn_error_report_t *report, int32_t instance) : _context(monitor, report, (uint32_t)instance), _env(_context), _instance(instance) {
  }

  __device__ __forceinline__ void fixed_window_powm_odd(bn_t &result, const bn_t &x, const bn_t &power, const bn_t &modulus) {
    bn_t       t;
    bn_local_t window[1<<window_bits];
    int32_t    index, position, offset;
    uint32_t   np0;

    // conmpute x^power mod modulus, using the fixed window algorithm
    // requires:  x<modulus,  modulus is odd

    // compute x^0 (in Montgomery space, this is just 2^BITS - modulus)
    cgbn_negate(_env, t, modulus);
    cgbn_store(_env, window+0, t);

    // convert x into Montgomery space, store into window table
    np0=cgbn_bn2mont(_env, result, x, modulus);
    cgbn_store(_env, window+1, result);
    cgbn_set(_env, t, result);

    // compute x^2, x^3, ... x^(2^window_bits-1), store into window table
    #pragma nounroll
    for(index=2;index<(1<<window_bits);index++) {
      cgbn_mont_mul(_env, result, result, t, modulus, np0);
      cgbn_store(_env, window+index, result);
    }

    // find leading high bit
    position=params::BITS - cgbn_clz(_env, power);

    // break the exponent into chunks, each window_bits in length
    // load the most significant non-zero exponent chunk
    offset=position % window_bits;
    if(offset==0)
      position=position-window_bits;
    else
      position=position-offset;
    index=cgbn_extract_bits_ui32(_env, power, position, window_bits);
    cgbn_load(_env, result, window+index);

    // process the remaining exponent chunks
    while(position>0) {
      // square the result window_bits times
      #pragma nounroll
      for(int sqr_count=0;sqr_count<window_bits;sqr_count++)
        cgbn_mont_sqr(_env, result, result, modulus, np0);

      // multiply by next exponent chunk
      position=position-window_bits;
      index=cgbn_extract_bits_ui32(_env, power, position, window_bits);
      cgbn_load(_env, t, window+index);
      cgbn_mont_mul(_env, result, result, t, modulus, np0);
    }

    // we've processed the exponent now, convert back to normal space
    cgbn_mont2bn(_env, result, result, modulus, np0);
  }

  __device__ __forceinline__ void sliding_window_powm_odd(bn_t &result, const bn_t &x, const bn_t &power, const bn_t &modulus) {
    bn_t         t, starts;
    int32_t      index, position, leading;
    uint32_t     mont_inv;
    bn_local_t   odd_powers[1<<window_bits-1];

    // conmpute x^power mod modulus, using Constant Length Non-Zero windows (CLNZ).
    // requires:  x<modulus,  modulus is odd

    // find the leading one in the power
    leading=params::BITS-1-cgbn_clz(_env, power);
    if(leading>=0) {
      // convert x into Montgomery space, store in the odd powers table
      mont_inv=cgbn_bn2mont(_env, result, x, modulus);

      // compute t=x^2 mod modulus
      cgbn_mont_sqr(_env, t, result, modulus, mont_inv);

      // compute odd powers window table: x^1, x^3, x^5, ...
      cgbn_store(_env, odd_powers, result);
      #pragma nounroll
      for(index=1;index<(1<<window_bits-1);index++) {
        cgbn_mont_mul(_env, result, result, t, modulus, mont_inv);
        cgbn_store(_env, odd_powers+index, result);
      }

      // starts contains an array of bits indicating the start of a window
      cgbn_set_ui32(_env, starts, 0);

      // organize p as a sequence of odd window indexes
      position=0;
      while(true) {
        if(cgbn_extract_bits_ui32(_env, power, position, 1)==0)
          position++;
        else {
          cgbn_insert_bits_ui32(_env, starts, starts, position, 1, 1);
          if(position+window_bits>leading)
            break;
          position=position+window_bits;
        }
      }

      // load first window.  Note, since the window index must be odd, we have to
      // divide it by two before indexing the window table.  Instead, we just don't
      // load the index LSB from power
      index=cgbn_extract_bits_ui32(_env, power, position+1, window_bits-1);
      cgbn_load(_env, result, odd_powers+index);
      position--;

      // Process remaining windows
      while(position>=0) {
        cgbn_mont_sqr(_env, result, result, modulus, mont_inv);
        if(cgbn_extract_bits_ui32(_env, starts, position, 1)==1) {
          // found a window, load the index
          index=cgbn_extract_bits_ui32(_env, power, position+1, window_bits-1);
          cgbn_load(_env, t, odd_powers+index);
          cgbn_mont_mul(_env, result, result, t, modulus, mont_inv);
        }
        position--;
      }

      // convert result from Montgomery space
      cgbn_mont2bn(_env, result, result, modulus, mont_inv);
    }
    else {
      // p=0, thus x^p mod modulus=1
      cgbn_set_ui32(_env, result, 1);
    }
  }

  // this is assuming power will equal 2^t. then we would have a lot of memory issues.
  __device__ __forceinline__ void grouped_fixed_window_powm_odd(bn_t &result, const bn_t &x, const uint32_t t, const bn_t &modulus, const uint32_t grouping) {

    // First we calculate the exponent, in this case 2^grouping.
    // Then we divide to get an index, and take the modulus to get the last exponent.
    bn_t primary_exponent;
    // this is 1 because 1 = 2^0, we shift grouping times so the result will be
    // 2^0 * 2^grouping = 2^(0 + grouping) = 2^grouping
    cgbn_set_ui32(_env, primary_exponent, 1);
    cgbn_shift_left(_env, primary_exponent, primary_exponent, grouping);

    bn_t two;
    cgbn_set_ui32(_env, two, 2);
    bn_t tee;
    cgbn_set_ui32(_env, tee, t);
    bn_t expon;
    fixed_window_powm_odd(expon, two, tee, modulus);
    fixed_window_powm_odd(result, x, expon, modulus);


    /* // limit = t / grouping */
    /* // we don't care about the result being stored in a bn_t */
    /* uint32_t limit = t / grouping; */

    /* // final_grouping = t % grouping */
    /* // we don't care about the result being stored in a bn_t */
    /* const uint32_t final_grouping = t % grouping; */

    /* bn_t one; */
    /* cgbn_set_ui32(_env, one, 1); */
    /* bn_t zero; */
    /* cgbn_set_ui32(_env, zero, 0); */

    /* // Now we take 2^final_grouping for the final exponent */
    /* bn_t final_exponent; */
    /* cgbn_set_ui32(_env, final_exponent, 1); */
    /* cgbn_shift_left(_env, final_exponent, final_exponent, final_grouping); */

    /* // x is not constant so we create a mutable one */
    /* bn_t mut_x; */
    /* cgbn_set(_env, mut_x, x); */
    /* // now we do this a bunch of times */
    /* while (limit > 0) { */
    /*   // x = x ^ primary_exponent (mod N) */
    /*   fixed_window_powm_odd(mut_x, mut_x, primary_exponent, modulus); */
    /*   limit--; */
    /* } */

    /* // and finally, the last will store the result */
    /* fixed_window_powm_odd(result, mut_x, final_exponent, modulus); */

    return;
  }

  // this is assuming power will equal 2^t. if we actually calculated 2^t then we would have a lot of memory issues.
  __device__ __forceinline__ void grouped_sliding_window_powm_odd(bn_t &result, const bn_t &x, const bn_t &t, const bn_t &modulus, const uint32_t grouping) {

    // First we calculate the exponent, in this case 2^grouping.
    // Then we divide to get an index, and take the modulus to get the last exponent.
    bn_t primary_exponent;
    // this is 1 because 1 = 2^0, we shift grouping times so the result will be
    // 2^0 * 2^grouping = 2^(0 + grouping) = 2^grouping
    cgbn_set_ui32(_env, primary_exponent, 1);
    cgbn_shift_left(_env, primary_exponent, primary_exponent, grouping);

    // limit = t / grouping
    // we don't care about the result being stored in a bn_t
    bn_t limit;
    cgbn_div_ui32(_env, limit, t, grouping);

    // final_grouping = t % grouping
    // we don't care about the result being stored in a bn_t
    const uint32_t final_grouping = cgbn_rem_ui32(_env, t, grouping);

    bn_t one;
    cgbn_set_ui32(_env, one, 1);
    bn_t zero;
    cgbn_set_ui32(_env, zero, 0);

    // Now we take 2^final_grouping for the final exponent
    bn_t final_exponent;
    cgbn_set_ui32(_env, final_exponent, 1);
    cgbn_shift_left(_env, final_exponent, final_exponent, final_grouping);

    // x is not constant so we create a mutable one
    bn_t mut_x;
    cgbn_set(_env, mut_x, x);
    // now we do this a bunch of times
    while (!cgbn_equals(_env, limit, zero)) {
      // x = x ^ primary_exponent (mod N)
      sliding_window_powm_odd(mut_x, mut_x, primary_exponent, modulus);
      cgbn_sub(_env, limit, limit, one);
    }

    // and finally, the last will store the result
    sliding_window_powm_odd(result, mut_x, final_exponent, modulus);

    return;
  }

  __host__ static instance_t *generate_instances(uint32_t count) {
    instance_t *instances=(instance_t *)malloc(sizeof(instance_t)*count);
    int         index;

    for(index=0;index<count;index++) {
      random_words(instances[index].x._limbs, params::BITS/32);
      random_words(instances[index].power._limbs, params::BITS/32);
      random_words(instances[index].modulus._limbs, params::BITS/32);

      // ensure modulus is odd
      instances[index].modulus._limbs[0] |= 1;

      // ensure modulus is greater than
      if(compare_words(instances[index].x._limbs, instances[index].modulus._limbs, params::BITS/32)>0) {
        swap_words(instances[index].x._limbs, instances[index].modulus._limbs, params::BITS/32);

        // modulus might now be even, ensure it's odd
        instances[index].modulus._limbs[0] |= 1;
      }
      else if(compare_words(instances[index].x._limbs, instances[index].modulus._limbs, params::BITS/32)==0) {
        // since modulus is odd and modulus = x, we can just subtract 1 from x
        instances[index].x._limbs[0] -= 1;
      }
    }
    return instances;
  }

  // this generates timelock puzzle instances, for example 2^2^4444 or 5^2^12345 or 7^2^3333
  // all mod the same N. This is the case because when testing, we're generating the instances
  // probably knowing the factorization of N.
  // This way we can test that a^2^t (mod N), calculated from cgbn, = a^(2^t (mod phi(N)) (mod N),
  // calculated from gmp.
  // Either that, or we just let GMP do all of the work
  __host__ static instance_t *generate_puzzle_instances(uint32_t count, const uint32_t t, const mpz_t N) {
    instance_t *instances=(instance_t *)malloc(sizeof(instance_t)*count);
    int         index;

    mpz_t two; // 2
    mpz_init(two);
    mpz_set_ui(two, 2);

    mpz_t five; // 5
    mpz_init(five);
    mpz_set_ui(five, 5);

    mpz_t seven; // 7
    mpz_init(seven);
    mpz_set_ui(seven, 7);

    mpz_t thirteen; // 13
    mpz_init(thirteen);
    mpz_set_ui(thirteen, 13);

    mpz_t maxval;
    mpz_init(maxval);

    for(index=0;index<count;index++) {
        // create 2^whatever
        mpz_t e;
        mpz_init(e);

        mpz_pow_ui(e, two, t);
        // just alternate between our bases
        switch (index % 4) {
          case 0:
            // base = 2
            instances[index] = create_instance(two, e, N);
          case 1:
            instances[index] = create_instance(five, e, N);
          case 2:
            instances[index] = create_instance(seven, e, N);
          case 3:
            instances[index] = create_instance(thirteen, e, N);
        }

        mpz_clear(e);
    }

    mpz_clear(maxval);
    mpz_clear(two);
    mpz_clear(five);
    mpz_clear(seven);
    mpz_clear(thirteen);
    return instances;
  }

  __host__ static instance_t create_instance(const mpz_t x, const mpz_t e, const mpz_t N) {
    instance_t instance;
    // first, we get the number of limbs

    const size_t num_limbs_x = mpz_size(x);
    const size_t num_limbs_e = mpz_size(e);
    const size_t num_limbs_N = mpz_size(N);

    // just have these assertions in case anyone tries to pull any funny business
    // any one of these failing means the input is too big and we would have caused a segfault
    // the size of instance x, e, N limbs should be params::BITS/32, so if these were not true
    // then we would loop over them and cause a segfault.
    assert(num_limbs_x <= params::BITS/32);
    assert(num_limbs_e <= params::BITS/32);
    assert(num_limbs_N <= params::BITS/32);

    // start with x
    for (int i = 0; i < num_limbs_x; ++i) {
        // get limb i, put in x.
        instance.x._limbs[i] = mpz_getlimbn(x, i);
    }

    // now do e
    for (int i = 0; i < num_limbs_e; ++i) {
        // get limb i, put in e.
        instance.power._limbs[i] = mpz_getlimbn(e, i);
    }

    // now do N
    for (int i = 0; i < num_limbs_N; ++i) {
        // get limb i, put in N.
        instance.modulus._limbs[i] = mpz_getlimbn(N, i);
    }

    // now we're going to assert that the modulus is odd, just because we don't want to create
    // instances that would be incorrect for powm_ODD_t.
    assert(instance.modulus._limbs[0] & 1);

    return instance;
  }

  __host__ static instance_t *create_instances(const mpz_t* xs, const mpz_t* es, const mpz_t* Ns, const uint32_t count) {
    instance_t *instances=(instance_t *)malloc(sizeof(instance_t)*count);
    int         index;

    for(index=0;index<count;index++) {
      instances[index] = create_instance(xs[index], es[index], Ns[index]);
    }
    return instances;
  }

  __host__ static void verify_results(instance_t *instances, uint32_t count) {
    mpz_t x, p, m, computed, correct;

    mpz_init(x);
    mpz_init(p);
    mpz_init(m);
    mpz_init(computed);
    mpz_init(correct);
    int wrong = 0;

    for(int index=0;index<count;index++) {
      to_mpz(x, instances[index].x._limbs, params::BITS/32);
      to_mpz(p, instances[index].power._limbs, params::BITS/32);
      to_mpz(m, instances[index].modulus._limbs, params::BITS/32);
      to_mpz(computed, instances[index].result._limbs, params::BITS/32);

      size_t instance_x_size = mpz_sizeinbase(x, 2);
      size_t instance_p_size = mpz_sizeinbase(p, 2);
      size_t instance_m_size = mpz_sizeinbase(m, 2);
      size_t instance_r_size = mpz_sizeinbase(computed, 2);
      /* printf("Instance %d: Number of bits in x: %lu\n", index, instance_x_size); */
      /* printf("Instance %d: Number of bits in p: %lu\n", index, instance_p_size); */
      /* printf("Instance %d: Number of bits in m: %lu\n", index, instance_m_size); */
      /* printf("Instance %d: Number of bits in r: %lu\n", index, instance_r_size); */
      mpz_powm(correct, x, p, m);
      if(mpz_cmp(correct, computed)!=0) {
        /* printf("gpu inverse kernel failed on instance %d\n", index); */
          wrong++;
        // return;
      }
    }
    printf("Number of powm's computed: %d\n", count);

    mpz_clear(x);
    mpz_clear(p);
    mpz_clear(m);
    mpz_clear(computed);
    mpz_clear(correct);

    if (wrong == 0) {
        printf("All results match\n");
    } else {
        printf("Not all results match, %d wrong\n", wrong);
    }
  }
};

// kernel implementation using cgbn
//
// Unfortunately, the kernel must be separate from the powm_odd_t class

template<class params>
__global__ void kernel_powm_odd(cgbn_error_report_t *report, typename powm_odd_t<params>::instance_t *instances, uint32_t count) {
  int32_t instance;

  // decode an instance number from the blockIdx and threadIdx
  instance=(blockIdx.x*blockDim.x + threadIdx.x)/params::TPI;
  if(instance>=count)
    return;

  powm_odd_t<params>                 po(cgbn_report_monitor, report, instance);
  typename powm_odd_t<params>::bn_t  r, x, p, m;

  // the loads and stores can go in the class, but it seems more natural to have them
  // here and to pass in and out bignums
  cgbn_load(po._env, x, &(instances[instance].x));
  cgbn_load(po._env, p, &(instances[instance].power));
  cgbn_load(po._env, m, &(instances[instance].modulus));

  // this can be either fixed_window_powm_odd or sliding_window_powm_odd.
  // when TPI<32, fixed window runs much faster because it is less divergent, so we use it here
  po.fixed_window_powm_odd(r, x, p, m);
  //   OR
  // po.sliding_window_powm_odd(r, x, p, m);

  cgbn_store(po._env, &(instances[instance].result), r);
}

// grouped fixed kernel implementation using cgbn -- IMPORTANT! default grouping = 1024
template<class params>
__global__ void grouped_fixed_kernel_powm_odd(cgbn_error_report_t *report, typename powm_odd_t<params>::instance_t *instances, const uint32_t count, const uint32_t grouping, const uint32_t time_value) {
  int32_t instance;

  // decode an instance number from the blockIdx and threadIdx
  instance=(blockIdx.x*blockDim.x + threadIdx.x)/params::TPI;
  if(instance>=count)
    return;

  powm_odd_t<params>                 po(cgbn_report_monitor, report, instance);
  typename powm_odd_t<params>::bn_t  r, x, p, m;

  // the loads and stores can go in the class, but it seems more natural to have them
  // here and to pass in and out bignums
  cgbn_load(po._env, x, &(instances[instance].x));
  cgbn_load(po._env, p, &(instances[instance].power));
  cgbn_load(po._env, m, &(instances[instance].modulus));

  // this can be either fixed_window_powm_odd or sliding_window_powm_odd.
  // when TPI<32, fixed window runs much faster because it is less divergent, so we use it here
  po.fixed_window_powm_odd(r, x, p, m);
  //   OR
  // po.grouped_sliding_window_powm_odd(r, x, p, m, grouping);

  cgbn_store(po._env, &(instances[instance].result), r);
}

// grouped sliding window kernel implementation using cgbn -- IMPORTANT! default grouping = 1024
template<class params>
__global__ void grouped_sliding_kernel_powm_odd(cgbn_error_report_t *report, typename powm_odd_t<params>::instance_t *instances, const uint32_t count, const uint32_t grouping) {
  int32_t instance;

  // decode an instance number from the blockIdx and threadIdx
  instance=(blockIdx.x*blockDim.x + threadIdx.x)/params::TPI;
  if(instance>=count)
    return;

  powm_odd_t<params>                 po(cgbn_report_monitor, report, instance);
  typename powm_odd_t<params>::bn_t  r, x, p, m;

  // the loads and stores can go in the class, but it seems more natural to have them
  // here and to pass in and out bignums
  cgbn_load(po._env, x, &(instances[instance].x));
  cgbn_load(po._env, p, &(instances[instance].power));
  cgbn_load(po._env, m, &(instances[instance].modulus));

  // this can be either fixed_window_powm_odd or sliding_window_powm_odd.
  // when TPI<32, fixed window runs much faster because it is less divergent, so we use it here
  // po.grouped_fixed_window_powm_odd(r, x, p, m, grouping);
  //   OR
  po.grouped_sliding_window_powm_odd(r, x, p, m, grouping);

  cgbn_store(po._env, &(instances[instance].result), r);
}


template<class params>
void run_test(uint32_t instance_count) {
  typedef typename powm_odd_t<params>::instance_t instance_t;

  instance_t          *instances, *gpuInstances;
  cgbn_error_report_t *report;
  int32_t              TPB=(params::TPB==0) ? 128 : params::TPB;    // default threads per block to 128
  int32_t              TPI=params::TPI, IPB=TPB/TPI;                // IPB is instances per block

  printf("Generating instances ...\n");
  instances=powm_odd_t<params>::generate_instances(instance_count);

  printf("Copying instances to the GPU ...\n");
  CUDA_CHECK(hipSetDevice(0));
  CUDA_CHECK(hipMalloc((void **)&gpuInstances, sizeof(instance_t)*instance_count));
  CUDA_CHECK(hipMemcpy(gpuInstances, instances, sizeof(instance_t)*instance_count, hipMemcpyHostToDevice));

  // create a cgbn_error_report for CGBN to report back errors
  CUDA_CHECK(cgbn_error_report_alloc(&report));

  printf("Running GPU kernel ...\n");

  // launch kernel with blocks=ceil(instance_count/IPB) and threads=TPB
  kernel_powm_odd<params><<<(instance_count+IPB-1)/IPB, TPB>>>(report, gpuInstances, instance_count);

  // error report uses managed memory, so we sync the device (or stream) and check for cgbn errors
  CUDA_CHECK(hipDeviceSynchronize());
  CGBN_CHECK(report);

  // copy the instances back from gpuMemory
  printf("Copying results back to CPU ...\n");
  CUDA_CHECK(hipMemcpy(instances, gpuInstances, sizeof(instance_t)*instance_count, hipMemcpyDeviceToHost));

  printf("Verifying the results ...\n");
  powm_odd_t<params>::verify_results(instances, instance_count);

  // clean up
  free(instances);
  CUDA_CHECK(hipFree(gpuInstances));
  CUDA_CHECK(cgbn_error_report_free(report));
}

template<class params>
void run_puzzle_test(const uint32_t instance_count, const uint32_t time_value) {
  typedef typename powm_odd_t<params>::instance_t instance_t;

  instance_t          *instances, *gpuInstances;
  cgbn_error_report_t *report;
  int32_t              TPB=(params::TPB==0) ? 128 : params::TPB;    // default threads per block to 128
  int32_t              TPI=params::TPI, IPB=TPB/TPI;                // IPB is instances per block

  printf("Generating composite to be used in puzzles...\n");

  // initialize private key
  private_key priv;
  mpz_init(priv.n);
  mpz_init(priv.e);
  mpz_init(priv.d);
  mpz_init(priv.p);
  mpz_init(priv.q);

  // initialize public key
  public_key  pub;
  mpz_init(pub.n);
  mpz_init(pub.e);

  // now generate the keys with 4096 bits
  generate_keys(&priv, &pub, 4096);

  printf("Generating puzzle instances ...\n");
  instances=powm_odd_t<params>::generate_puzzle_instances(instance_count, time_value, priv.n);

  printf("Copying instances to the GPU ...\n");
  CUDA_CHECK(hipSetDevice(0));
  CUDA_CHECK(hipMalloc((void **)&gpuInstances, sizeof(instance_t)*instance_count));
  CUDA_CHECK(hipMemcpy(gpuInstances, instances, sizeof(instance_t)*instance_count, hipMemcpyHostToDevice));

  // create a cgbn_error_report for CGBN to report back errors
  CUDA_CHECK(cgbn_error_report_alloc(&report));

  // declaring argument of time()
  time_t my_time = time(NULL);
  // ctime() used to give the present time
  printf("Start Time: %s", ctime(&my_time));

  printf("Running GPU kernel ...\n");

  // launch kernel with blocks=ceil(instance_count/IPB) and threads=TPB
  kernel_powm_odd<params><<<(instance_count+IPB-1)/IPB, TPB>>>(report, gpuInstances, instance_count);

  // error report uses managed memory, so we sync the device (or stream) and check for cgbn errors
  CUDA_CHECK(hipDeviceSynchronize());
  CGBN_CHECK(report);

  // declaring argument of time()
  my_time = time(NULL);
  // ctime() used to give the present time
  printf("Stop Time: %s", ctime(&my_time));

  // copy the instances back from gpuMemory
  printf("Copying results back to CPU ...\n");
  CUDA_CHECK(hipMemcpy(instances, gpuInstances, sizeof(instance_t)*instance_count, hipMemcpyDeviceToHost));

  // declaring argument of time()
  my_time = time(NULL);
  // ctime() used to give the present time
  printf("Verify Start Time: %s", ctime(&my_time));

  printf("Verifying the results ...\n");
  powm_odd_t<params>::verify_results(instances, instance_count);

  // declaring argument of time()
  my_time = time(NULL);
  // ctime() used to give the present time
  printf("Verify Stop Time: %s", ctime(&my_time));

  // clean up
  free(instances);

  // clear private key
  mpz_clear(priv.n);
  mpz_clear(priv.e);
  mpz_clear(priv.d);
  mpz_clear(priv.p);
  mpz_clear(priv.q);

  // clear public key
  mpz_clear(pub.n);
  mpz_clear(pub.e);

  CUDA_CHECK(hipFree(gpuInstances));
  CUDA_CHECK(cgbn_error_report_free(report));
}

/* int main() { */
/*   typedef powm_params_t<8, 1024, 5> params; */

/*   run_test<params>(10000); */
/* } */
